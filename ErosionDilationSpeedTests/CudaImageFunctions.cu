#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include "CudaImageFunctions.cuh"

#include <stdio.h>
#include <algorithm>

// Kernel for erosion
__global__ void erosionKernel(int rows, int columns, const unsigned char* input, unsigned char* output)
{
   // Calculate row and column index for the current thread
   int r = blockIdx.y * blockDim.y + threadIdx.y;
   int c = blockIdx.x * blockDim.x + threadIdx.x;

   // Still ignoring the edges
   if (r > 0 && r < rows - 1 && c > 0 && c < columns - 1) {
      unsigned char minValue = 255;

      // little loop through the surrounding 9 pixels
      for (int r_local = r - 1; r_local <= r + 1; r_local++) {
         for (int c_local = c - 1; c_local <= c + 1; c_local++) {
            if (input[r_local * columns + c_local] < minValue) {
               minValue = input[r_local * columns + c_local];
            }
         }
      }

      output[r * columns + c] = minValue;
   }
}

// Kernel for dilation
__global__ void dilationKernel(int rows, int columns, const unsigned char* input, unsigned char* output)
{
   // Calculate row and column index for the current thread
   int r = blockIdx.y * blockDim.y + threadIdx.y;
   int c = blockIdx.x * blockDim.x + threadIdx.x;

   // Still ignoring the edges
   if (r > 0 && r < rows - 1 && c > 0 && c < columns - 1) {
      unsigned char maxValue = 0;

      // little loop through the surrounding 9 pixels
      for (int r_local = r - 1; r_local <= r + 1; r_local++) {
         for (int c_local = c - 1; c_local <= c + 1; c_local++) {
            if (input[r_local * columns + c_local] > maxValue) {
               maxValue = input[r_local * columns + c_local];
            }
         }
      }

      output[r * columns + c] = maxValue;
   }
}


void CudaImageFunctions::PerformErosionCUDA(int rows, int columns, const unsigned char* input, unsigned char* output)
{
   size_t imageSize = rows * columns * sizeof(unsigned char);

   unsigned char* d_input = nullptr;
   unsigned char* d_output = nullptr;

   // Allocate memory on the GPU
   hipMalloc((void**)&d_input, imageSize);
   hipMalloc((void**)&d_output, imageSize);

   // Copy the input data from host to device
   hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);

   // Define thread block and grid sizes
   dim3 blockSize(16, 16); // Each block has 16x16 threads
   dim3 gridSize((columns + blockSize.x - 1) / blockSize.x,
      (rows + blockSize.y - 1) / blockSize.y);

   // Launch the kernel
   erosionKernel << <gridSize, blockSize >> > (rows, columns, d_input, d_output);

   // Wait for the kernel to finish
   hipDeviceSynchronize();

   // Copy the result back to the host
   hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

   // Free GPU memory
   hipFree(d_input);
   hipFree(d_output);
}


void CudaImageFunctions::PerformDilationCUDA(int rows, int columns, const unsigned char* input, unsigned char* output)
{
   size_t imageSize = rows * columns * sizeof(unsigned char);

   unsigned char* d_input = nullptr;
   unsigned char* d_output = nullptr;

   // Allocate memory on the GPU
   hipMalloc((void**)&d_input, imageSize);
   hipMalloc((void**)&d_output, imageSize);

   // Copy the input data from host to device
   hipMemcpy(d_input, input, imageSize, hipMemcpyHostToDevice);

   // Define thread block and grid sizes
   dim3 blockSize(16, 16); // Each block has 16x16 threads
   dim3 gridSize((columns + blockSize.x - 1) / blockSize.x,
      (rows + blockSize.y - 1) / blockSize.y);

   // Launch the kernel
   dilationKernel << <gridSize, blockSize >> > (rows, columns, d_input, d_output);

   // Wait for the kernel to finish
   hipDeviceSynchronize();

   // Copy the result back to the host
   hipMemcpy(output, d_output, imageSize, hipMemcpyDeviceToHost);

   // Free GPU memory
   hipFree(d_input);
   hipFree(d_output);
}
